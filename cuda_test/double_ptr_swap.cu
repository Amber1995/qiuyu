#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__device__ int block_idx(int grid_dim) {
  int block_id = blockIdx.x + (grid_dim == 2 ? 1 : 0) * blockIdx.y * gridDim.x +
                 (grid_dim == 3 ? 1 : 0) * blockIdx.z * gridDim.x * gridDim.y;
  return block_id;
}

__device__ int thread_idx(int grid_dim, int block_dim) {
  // ! thread id inside a block
  unsigned long int threadIdInBlock =
      threadIdx.x + (block_dim == 2 ? 1 : 0) * threadIdx.y * blockDim.x +
      (block_dim == 3 ? 1 : 0) * threadIdx.z * blockDim.x * blockDim.z;
  //! block id
  unsigned long int block_id = block_idx(grid_dim);
  // ! block size
  unsigned long int threadsPerblock = blockDim.x *
                                      (block_dim == 2 ? blockDim.y : 1) *
                                      (block_dim == 3 ? blockDim.z : 1);
  unsigned long int thread_id = block_id * threadsPerblock + threadIdInBlock;
  return thread_id;
}

void show_array(int** array, int nbrows, int nbcols) {
  for (int i = 0; i < nbrows; ++i) {
    for (int j = 0; j < nbcols; ++j) std::cout << array[i][j] << std::endl;
  }
}

__global__ void gpu_array_swap(int** ptr_gpu, int nbrows, int nbcols,
                               int grid_dim, int block_dim) {
  int thread_id = thread_idx(grid_dim, block_dim);
  int i = floorf((thread_id + 1) / nbcols);
  int j = thread_id + 1 - i * nbcols;
  // To test if kernel function runs
  //(Fact:it runs)
  printf("I am running!\n");
  //__syncthreads();

  // To see if ptr_gpu has been transferred
  //(Fact: Not transferred in and the printf command below didn't run)
  ptr_gpu[1][1] = 100;
  printf("%d %d\n", ptr_gpu[1][1], ptr_gpu[i][j]);
}

int main() {
  //! Declare a double pointer on the host
  int **ptr_cpu = NULL, **ptr_gpu = NULL;
  const int nbcols = 4, nbrows = 5;
  int N = nbrows * nbcols;
  ptr_cpu = new int*[nbrows];
  for (int i = 0; i < nbrows; i++) ptr_cpu[i] = new int[nbcols];

  int k = 0;
  for (int i = 0; i < nbrows; ++i) {
    for (int j = 0; j < nbcols; ++j) ptr_cpu[i][j] = k++;
  };

  //! Allocate memory on GPU
  hipMalloc(&ptr_gpu, nbrows * sizeof(int*));
  hipMemcpy(ptr_gpu, ptr_cpu, nbrows * sizeof(int*), hipMemcpyHostToDevice);
  if (ptr_gpu == NULL) {
    printf("Couldn't allocate GPU memory\n");
  };

  // 2d block: threads in x and y directions
  dim3 bs = dim3(nbrows, nbcols);
  // 1d grid: block in x direction
  int grid = 1;  // Only 1 block
  // grid and block dimensions
  int grid_dim = 1, block_dim = 2;

  // ! Call kernel function on a 1d-grid 2d-block
  gpu_array_swap<<<grid, bs>>>(ptr_gpu, nbrows, nbcols, grid_dim, block_dim);

  //! Copy data from device to host
  hipDeviceSynchronize();
  hipMemcpy(ptr_cpu, ptr_gpu, nbrows * sizeof(int*), hipMemcpyDeviceToHost);
  
  //  show_array(ptr_cpu, nbrows, nbcols);

  hipFree(ptr_gpu);
  free(ptr_cpu);
}
